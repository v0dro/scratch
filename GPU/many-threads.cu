
#include <hip/hip_runtime.h>
#include <stdio.h>
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

const int blocksize = 1024;
const int factor = 16;
const int arraysize = blocksize / factor;

template <typename T>
__global__ void addition_test_kernel(T * a, T * sum) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int idx = (tid) % arraysize;

  sum[idx] += a[idx];

  atomicAdd(&sum[idx], a[idx]);

  if (idx == 2)
    printf("%d %d %d %d -> ", idx, tid, sum[idx], a[idx]);
}

template <typename T>
void test_atomic_int() {
  dim3 dimBlock(blocksize, 1);
  dim3 dimGrid(1, 1);

  T *a, *b, *sum, *answer, *ad, *bd, *sumd, *answerd;
  
  a = (T*)malloc(arraysize * sizeof(T));
  sum = (T*)malloc(arraysize * sizeof(T));
  answer = (T*)malloc(arraysize * sizeof(T));

  for (int i = 0; i < arraysize; ++i) {
    a[i] = 1;
    sum[i] = 0;
    answer[i] = i + i;
  }

  hipMalloc((void**)&ad, arraysize * sizeof(T));
  gpuErrchk(hipPeekAtLastError());
  hipMalloc((void**)&sumd, arraysize * sizeof(T));
  gpuErrchk(hipPeekAtLastError());
  
  hipMemcpy(ad, a, arraysize * sizeof(T), hipMemcpyHostToDevice);
  gpuErrchk(hipGetLastError());
    
  hipMemcpy(sumd, sum, arraysize * sizeof(T), hipMemcpyHostToDevice);
  gpuErrchk(hipPeekAtLastError());

  printf("addition kernel;\n");
  addition_test_kernel<<<dimGrid, dimBlock>>>(ad, sumd);
  gpuErrchk(hipPeekAtLastError());
  hipMemcpy(sum, sumd, arraysize * sizeof(T), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  for (int i = 0; i < arraysize; ++i) {

    printf("sum[%d]: %d\n", i, sum[i]);    
  }

}

int main(int argc, char *argv[])
{
  test_atomic_int<int32_t>();
  return 0;
}
