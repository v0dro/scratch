
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

const int blocksize = 1;

typedef unsigned char uint40[5];

struct alignas(2) BFloat16 {
  uint16_t x;
  BFloat16() = default;

  // constexpr C10_HOST_DEVICE BFloat16(unsigned short bits, from_bits_t) : x(bits){};
};

__global__
void check_size() {
  uint40 a;
  BFloat16 b16;
  
  printf("sizeof(unsigned int): %d\n", sizeof(unsigned int));
  printf("sizeof(char): %d\n", sizeof(char));
  printf("sizeof(size_t): %d\n", sizeof(size_t));
  printf("sizeof(unsigned long long int): %d\n", sizeof(unsigned long long int));
  printf("sizeof(double): %d\n", sizeof(double));
  printf("sizeof(uint40 a): %d\n", sizeof(a));
  printf("sizeof(int): %d\n", sizeof(int));
  printf("sizeof(unsigned short int): %d\n", sizeof(unsigned short int));

  printf("\n---------------------\n");
  printf("sizeof(unsigned int*): %d\n", sizeof(unsigned int*));
  printf("sizeof(char*): %d\n", sizeof(char*));
  printf("sizeof(size_t*): %d\n", sizeof(size_t*));
  printf("sizeof(unsigned long long int*): %d\n", sizeof(unsigned long long int*));
  printf("sizeof(double*): %d\n", sizeof(double*));
  printf("sizeof(uint40*): %d\n", sizeof(uint40*));
  printf("sizeof(BFloat16*): %d\n", sizeof(BFloat16*));
  printf("sizeof(int*): %d\n", sizeof(int*));
  printf("sizeof(unsigned short int*): %d\n", sizeof(unsigned short int*));

  
  printf("\n---------------------\n");
  printf("a: %ld, a+1: %ld\n", &a, &a+1);
  printf("&b16: %ld, &b16+1: %ld\n", &b16, &b16+1);
}

int main(int argc, char *argv[])
{
  dim3 dimBlock(blocksize, 1);
  dim3 dimGrid(1,1);

  check_size<<<dimGrid, dimBlock>>>();
  hipDeviceSynchronize(); 
  return 0;
}
